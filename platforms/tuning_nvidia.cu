#include "hip/hip_runtime.h"
#include "tuning_nvidia.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void gpuProcessKernel(const float* src, float* dst, float val, int len, Operation op) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        switch (op) {
            case COPY:
                dst[idx] = src[idx];
                break;
            case ZERO:
                dst[idx] = 0.0f;
                break;
            case ADD:
                dst[idx] += src[idx];
                break;
            case MULC:
                dst[idx] *= val;
                break;
        }
    }
}

void gpuProcess_32f(const float* src, float* dst, float val, int len, Operation op) {
    float* d_src = nullptr;
    float* d_dst = nullptr;

    // Cấp phát bộ nhớ trên GPU
    hipError_t err = hipMalloc(&d_src, len * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc d_src failed: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_dst, len * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc d_dst failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_src);
        return;
    }

    // Copy dữ liệu từ CPU -> GPU
    if (src != nullptr) {
        err = hipMemcpy(d_src, src, len * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "CUDA memcpy HostToDevice failed: " << hipGetErrorString(err) << std::endl;
            hipFree(d_src);
            hipFree(d_dst);
            return;
        }
    }

    // Gọi kernel để thực hiện phép toán
    int threadsPerBlock = 256;
    int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
    gpuProcessKernel<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_dst , val, len, op);
    err = hipGetLastError(); // Kiểm tra lỗi khi chạy kernel
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_src);
        hipFree(d_dst);
        return;
    }

    // Copy dữ liệu từ GPU -> CPU
    err = hipMemcpy(dst, d_dst, len * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy DeviceToHost failed: " << hipGetErrorString(err) << std::endl;
    }

    // Giải phóng bộ nhớ
    hipFree(d_src);
    hipFree(d_dst);
}
